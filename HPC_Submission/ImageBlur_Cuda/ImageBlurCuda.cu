#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include "lodepng.h"

//compile : nvcc -Wno-deprecated-gpu-targets -o ImageBlur ImageBlurCuda.cu lodepng.cpp  or just use make
// Execute :./ImageBlur

	
/**
 * Program takes a PNG image type as input then gives out a blured image of hte original image in the same location 
 * as blurImage.png image.
 * 
 * ImageBlur function is called by HOST and run on DEVICE. This function is responsible for making hte blur image of the user given image.
 * the function takes 5 parameter that includes the data to be processed and the pointer to save the new blur data along with the 
 * width and height of the image and by how much to blurl the original image. 
**/



__global__ void ImageBlur(unsigned char * imageOutPutDevice, unsigned char * imageDataHost,unsigned int w,unsigned int h, int blurAmount){

//initialization 
	int red = 0;
	int green = 0;
	int blue = 0;
	int a = 0;
	int x,y;


	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int pixel = idx*4;
    for(x = (pixel - (4 * blurAmount)); x <=  (pixel + (4 * blurAmount)); x+=4){
		if ((x > 0) && x < (h * w * 4) && ((x-4)/(4*w) == pixel/(4*w))){
			for(y = (x - (16 * w * blurAmount)); y <=  (x + (16 *w * blurAmount)); y+=(4*w)){
				if(y > 0 && y < ((h * w * 4))){
					red += imageDataHost[y];
					green += imageDataHost[1+y];
					blue += imageDataHost[2+y]; 
					a++;
				}
			}
		}
	}

	imageOutPutDevice[pixel] = red / a;
	imageOutPutDevice[1+pixel] = green / a;
	imageOutPutDevice[2+pixel] = blue / a;
	imageOutPutDevice[3+pixel] = imageDataHost[3+pixel];
}
// to check for errors during cudaFunctions
void CheckForError(hipError_t error){
     if (error)
    {
        printf("Something went wrong!\n");
        printf("Error: %s\n",hipGetErrorString(error));

        exit(1);
    }
}

int main(int argc, char **argv){
    hipError_t hipError_t; // to check for error during cuda functions.
    int blurAmount = 1; // To Determine the blur effects of picture
	unsigned int error; // to check for error during encoding and decoding the image data.
	unsigned char* img;
	unsigned int imageWidth;
	unsigned int imageHeight;
	const char* userInputImageName = "originalImage.png";  // image given by the user.
	const char* programOutputImage = "blurImageOutput.png";  // blur image created by the program.

	error = lodepng_decode32_file(&img, &imageWidth, &imageHeight, userInputImageName);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	int ARRAY_SIZE = imageWidth*imageHeight*4;
	int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imgInput[ARRAY_SIZE * 4];
	unsigned char host_imgOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imgInput[i] = img[i];
	}

	unsigned char * imageDataDevice;
	unsigned char * imageDataHost;
	

// Device memory
	hipError_t = hipMalloc((void**) &imageDataDevice, ARRAY_BYTES);
    CheckForError(hipError_t);

	hipError_t = hipMalloc((void**) &imageDataHost, ARRAY_BYTES);
    CheckForError(hipError_t);

	hipError_t = hipMemcpy(imageDataDevice, host_imgInput, ARRAY_BYTES, hipMemcpyHostToDevice);
    CheckForError(hipError_t);

	// Running the GPU funtion
	ImageBlur<<<imageHeight, imageWidth>>>(imageDataHost, imageDataDevice, imageWidth, imageHeight,blurAmount);

	hipError_t = hipMemcpy(host_imgOutput, imageDataHost, ARRAY_BYTES, hipMemcpyDeviceToHost);
    CheckForError(hipError_t);

	printf("The blur Image has been created!!\n");
	
	error = lodepng_encode32_file(programOutputImage, host_imgOutput, imageWidth, imageHeight);
	if(error){
		printf("error occured %u: %s\n", error, lodepng_error_text(error));
	}

	// free dynamic memory
	hipError_t = hipFree(imageDataDevice);
    CheckForError(hipError_t);
	hipError_t = hipFree(imageDataHost);
    CheckForError(hipError_t);

	return 0;
}
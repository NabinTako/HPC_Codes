#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"

/*****************

    command to compile
	nvcc -Wno-deprecated-gpu-targets -o ImageBlur ImageBlur.cu lodepng.cpp 
    // run - ./ImageBlur

 *****************/



__global__ void ImageBlur(unsigned char * imageOutPutDevice, unsigned char * imageDataHost,unsigned int w,unsigned int h, int blurAmount){

//initialization 
	int red = 0;
	int green = 0;
	int blue = 0;
	int x,y;
	int C = 0;


	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int pixel = idx*4;
    for(x = (pixel - (4 * blurAmount)); x <=  (pixel + (4 * blurAmount)); x+=4){
		if ((x > 0) && x < (h * w * 4) && ((x-4)/(4*w) == pixel/(4*w))){
			for(y = (x - (16 * w * blurAmount)); y <=  (x + (16 *w * blurAmount)); y+=(4*w)){
				if(y > 0 && y < ((h * w * 4))){
					red += imageDataHost[y];
					green += imageDataHost[1+y];
					blue += imageDataHost[2+y]; 
					C++;
				}
			}
		}
	}

	imageOutPutDevice[pixel] = red / C;
	imageOutPutDevice[1+pixel] = green / C;
	imageOutPutDevice[2+pixel] = blue / C;
	imageOutPutDevice[3+pixel] = imageDataHost[3+pixel];
}
// to check for errors during cudaFunctions
void CheckForError(hipError_t error){
     if (error)
    {
        printf("Something went wrong!\n");
        printf("Error: %s\n",hipGetErrorString(error));

        exit(1);
    }
}

int main(int argc, char **argv){
    hipError_t hipError_t;
    int blurAmount = 1; // To Determine the blur effects of picture
	unsigned int error;
	unsigned char* img;
	unsigned int imageWidth;
	unsigned int imageHeight;
	const char* userInputImageName = "test.png";  //the image given by user
	const char* programOutputImage = "testBlur.png";//the blur image created by program

	error = lodepng_decode32_file(&img, &imageWidth, &imageHeight, userInputImageName);
	if(error){
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	int ARRAY_SIZE = imageWidth*imageHeight*4;
	int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char host_imgInput[ARRAY_SIZE * 4];
	unsigned char host_imgOutput[ARRAY_SIZE * 4];

	for (int i = 0; i < ARRAY_SIZE; i++) {
		host_imgInput[i] = img[i];
	}

	unsigned char * imageDataDevice;
	unsigned char * imageDataHost;
	

// Device memory
	hipError_t = hipMalloc((void**) &imageDataDevice, ARRAY_BYTES);
    CheckForError(hipError_t);
	hipError_t = hipMalloc((void**) &imageDataHost, ARRAY_BYTES);
    CheckForError(hipError_t);



	hipError_t = hipMemcpy(imageDataDevice, host_imgInput, ARRAY_BYTES, hipMemcpyHostToDevice);
    CheckForError(hipError_t);


	ImageBlur<<<imageHeight, imageWidth>>>(imageDataHost, imageDataDevice, imageWidth, imageHeight,blurAmount);

	hipError_t = hipMemcpy(host_imgOutput, imageDataHost, ARRAY_BYTES, hipMemcpyDeviceToHost);
    CheckForError(hipError_t);

	printf("The blur Image has been created!\n");
	
	error = lodepng_encode32_file(programOutputImage, host_imgOutput, imageWidth, imageHeight);
	if(error){
		printf("error occured %u: %s\n", error, lodepng_error_text(error));
	}

// free dynamic memory
	hipError_t = hipFree(imageDataDevice);
    CheckForError(hipError_t);
	hipError_t = hipFree(imageDataHost);
    CheckForError(hipError_t);

	return 0;
}
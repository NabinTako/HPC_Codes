#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include <crypt.h>
#include "time_diff.h"

// compile the program by, nvcc -Wno-deprecated-gpu-targets -o crackPassword CrackPassword.cu time_diff.c -lrt
// or simply use make

struct timespec start, finish;
long long int difference;

__global__ void CrackPassword(char *encryptedPass,int* passwordFound)
{

    int x, y, z;                 // loop counters
    char salt[11];               // String used in hashing the password.
    char passwordCombination[7]; // The combination of letters currently being checked
    char *enc;
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // finding the threidId.
    memcpy(salt, encryptedPass + 0, sizeof(salt));
    salt[10] = '\0';

    for (x = 'A' + threadId; x <= 'Z'; x += blockDim.x * gridDim.x)
    {

        for (y = 'A'; y <= 'Z'; y++)
        {

            for (z = 0; z <= 99; z++)
            {

                printf(passwordCombination, sizeof(passwordCombination), "%c%c%02d", x, y, z);
                enc = (char *)crypt(passwordCombination, salt);

                if (memcmp(encryptedPass, enc, 97) == 0)
                {
                    *passwordFound = 1; // set the flag variable to tell other threads that password is found and break the operation.
                    break; // Break out of the innermost loop
                }
            }

            if (*passwordFound == 1)
            {
                break;
            }
        }

        if (*passwordFound == 1)
        {
            break;
        }
    }

    memcpy(encryptedPass, passwordCombination + 0, sizeof(passwordCombination));
    encryptedPass[10] = '\0';
}

int main()
{
    int blockNumber, threadNumber;
    printf("Number of Blocks to use: ");
    scanf("%d", &blockNumber);
    printf("Number of Threads per Block: ");
    scanf("%d", &threadNumber); // get the number of threads to use

    clock_gettime(CLOCK_MONOTONIC, &start);
    hipError_t error;

    // a shared flag to terminate the program when the password combination is found.
    int passwordFoundHost = 0;
    int *passwordFoundDevice;

    char *encryptedPasswordDevice;

    char encryptedPasswordHost[97] = "$6$WhyN0t$YHWmjxJ49Ob0xHhxZQuYsaJB5V8uukKIUYFnO.RSWKZOCB2H/i28hsPa2ibDXVicSHwAZUAkREqpc2P3066Fr1";

    error = hipMalloc(&encryptedPasswordDevice, sizeof(encryptedPasswordHost));
    if (error)
    {
        fprintf(stderr, "hipMalloc on encryptedPasswordDevice returned %d %s\n", error,
                hipGetErrorString(error));
        exit(1);
    }

    error = hipMemcpy(encryptedPasswordDevice, &encryptedPasswordHost, sizeof(encryptedPasswordHost), hipMemcpyHostToDevice);
    if (error)
    {
        fprintf(stderr, "hipMemcpy to encryptedPasswordDevice returned %d %s\n", error,
                hipGetErrorString(error));
    }

    // Copying the value of passwordfoundhost to be used in device.
    error = hipMalloc(&passwordFoundDevice, sizeof(int));
    if (error)
    {
        fprintf(stderr, "hipMalloc on encryptedPasswordDevice returned %d %s\n", error,
                hipGetErrorString(error));
        exit(1);
    }

    error = hipMemcpy(passwordFoundDevice, &passwordFoundHost, sizeof(int), hipMemcpyHostToDevice);
    if (error)
    {
        fprintf(stderr, "hipMemcpy to encryptedPasswordDevice returned %d %s\n", error,
                hipGetErrorString(error));
    }

    CrackPassword<<<blockNumber, threadNumber>>>(encryptedPasswordDevice,passwordFoundDevice);
    hipDeviceSynchronize();

    error = hipMemcpy(&passwordFoundHost, passwordFoundDevice, sizeof(int), hipMemcpyDeviceToHost);
    if (error)
    {
        fprintf(stderr, "hipMemcpy to passwordFoundHost returned %d %s\n", error,
        hipGetErrorString(error));
    }
    if(passwordFoundHost == 1){
        error = hipMemcpy(encryptedPasswordHost, encryptedPasswordDevice, sizeof(encryptedPasswordHost), hipMemcpyDeviceToHost);
        if (error)
        {
            fprintf(stderr, "hipMemcpy to encryptedPasswordHost returned %d %s\n", error,
            hipGetErrorString(error));
        }
    }else{
        printf("Password not found!!!\n");
    }

    // free the device pointers
    error = hipFree(encryptedPasswordDevice);
    if (error)
    {
        fprintf(stderr, "hipFree on encryptedPasswordDevice returned %d %s\n", error,
        hipGetErrorString(error));
        exit(1);
    }
    error = hipFree(passwordFoundDevice);
    if (error)
    {
        fprintf(stderr, "hipFree on passwordFoundDevice returned %d %s\n", error,
        hipGetErrorString(error));
        exit(1);
    }

    printf("result: encryptedPasswordHost = %s\n", encryptedPasswordHost);

    clock_gettime(CLOCK_MONOTONIC, &finish);

    // canculating the time taken by the program to find the unique password.
    time_difference(&start, &finish, &difference);

    printf("program Run Time %9.5lfs\n", difference / 1000000000.0);
    return 0;
}

// to calculate the total run time of the program
int time_difference(struct timespec *start, struct timespec *finish, long long int *difference)
{
    long long int ds = finish->tv_sec - start->tv_sec;
    long long int dn = finish->tv_nsec - start->tv_nsec;

    if (dn < 0)
    {
        ds--;
        dn += 1000000000;
    }
    *difference = ds * 1000000000 + dn;

    return !(*difference > 0);
}
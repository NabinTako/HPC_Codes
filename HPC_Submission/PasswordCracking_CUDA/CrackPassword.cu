#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

// Compile: nvcc -Wno-deprecated-gpu-targets -o crackPassword CrackPassword.cu -lrt  or just use make
// Execute: ./crackPassword

/** 
 * The Program Cracks password combinations of <smallalphabet><smallalphabet><number><number>. Example: aa12,nt99,wc23 etc.
 * The Program uses the power of GPU to run many threads parallelly n GPU. CPU first transferres data using the cudaRunTme library,
 * also while transferring the data, it also checks for errors whle transferring the datas in the dynamic memories using 
 * CheckForErrors function that takes type hipError_t (a cuda error type) as a parameter and displays the error messages.
 * 
 * The man GPU function to decryptPassword is run by CPU thats why it has __global__ attribute and other functoin 
 * with __device__ attribute in run by the GPU function. Among the gpu run functions, 
 * CudaCrypt is used to encrypt the newly generated plain password then be passed as parameter to function ComparePassword where
 * the newly encryptedPassword is compared with the userGivenEncryptedPassword and is true the function returns 1. 
 * If the comparePassword Function returns one CopyDecryptedPassword function is called by the main function of GPU which
 * copies the data of plainPassword and stores t on the gpu assigned pointer variable. 
 * The data stored in Assigned pointer variable is then copied back to hte CPU pointer data usng hipMemcpy function And dsplayed to the user.
**/

// Variables to track program run time
struct timespec start, finish;
long long int difference;

// Functions To be run on Device/ GPU.
__device__ char * CopyDecryptedPassword(char *dest,  char *src){
  int i = 0;
  do {
    dest[i] = src[i];}
  while (src[i++] != 0);
  return dest;
}
// compares the given two character array adn returns either true or false.
__device__ int ComparePassword( char *str_a,  char *str_b, unsigned len = 255){
	int numberOfCharactermatched = 0;
	int i = 0;
	int outOfRange = 0;
	while ((i < len) && !outOfRange) {
		if ((str_a[i] == 0) || (str_b[i] == 0)) {
			outOfRange = 1;
		}
		else if (str_a[i] == str_b[i]) {
			numberOfCharactermatched++;
		}
		i++;
	}
// as numberOfCharactermatched is increased every loop when the password character matches 
// and loop runs extra one time, so if match equals the number of loop minus 1, 
// every password characters matched as variable numberOfCharactermatched is increased by 1 in every loop except the last one.

	if(numberOfCharactermatched == i-1){ 
		return 1;
	}else{
		return 0;
	}

}

// used to encrypt the plaintext to encryptedText inside GPU.
__device__ char* CudaCrypt(char* passwordToEncrypt){

	char * encryptedPassword = (char *) malloc(sizeof(char) * 11);

    int z_AsciiValue = 122; // As ASCii value of Z is 90.
    int a_AsciiValue = 97; // As ASCii value of A is 65.
 
	encryptedPassword[0] = passwordToEncrypt[0] + 2;
	encryptedPassword[1] = passwordToEncrypt[0] - 2;
	encryptedPassword[2] = passwordToEncrypt[0] + 1;
	encryptedPassword[3] = passwordToEncrypt[1] + 3;
	encryptedPassword[4] = passwordToEncrypt[1] - 3;
	encryptedPassword[5] = passwordToEncrypt[1] - 1;
	encryptedPassword[6] = passwordToEncrypt[2] + 2;
	encryptedPassword[7] = passwordToEncrypt[2] - 2;
	encryptedPassword[8] = passwordToEncrypt[3] + 4;
	encryptedPassword[9] = passwordToEncrypt[3] - 4;
	encryptedPassword[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all lower case letter limits
			if(encryptedPassword[i] > z_AsciiValue){
				encryptedPassword[i] = (encryptedPassword[i] - z_AsciiValue) + a_AsciiValue;
			}else if(encryptedPassword[i] < a_AsciiValue){
				encryptedPassword[i] = (a_AsciiValue - encryptedPassword[i]) + a_AsciiValue;
			}
		}else{ //checking number section
			if(encryptedPassword[i] > 57){
				encryptedPassword[i] = (encryptedPassword[i] - 57) + 48;
			}else if(encryptedPassword[i] < 48){
				encryptedPassword[i] = (48 - encryptedPassword[i]) + 48;
			}
		}
	}
	return encryptedPassword;
}

// Called by the HOST/ CPU and run nsde the GPU/DEVICE.
// Used to Crack  the encrypted password given by the user.
__global__ void CrackPassword(char * alphabet, char * numbers, char * encryptedPassword) {

	char rawPasswordToCheck[4];

	rawPasswordToCheck[0] = alphabet[blockIdx.x];
	rawPasswordToCheck[1] = alphabet[blockIdx.y];

	rawPasswordToCheck[2] = numbers[threadIdx.x];
	rawPasswordToCheck[3] = numbers[threadIdx.y];


	if (ComparePassword(CudaCrypt(rawPasswordToCheck), encryptedPassword) == 1) {
		CopyDecryptedPassword(encryptedPassword, rawPasswordToCheck);
	}
}

// Functions To be run on Host/ CPU.
// to calculate the program total run time
int time_difference(struct timespec *start, struct timespec *finish, long long int *difference) {


  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  
  return !(*difference > 0);
}

void CheckForError(hipError_t error){
     if (error)
    {
        printf("Something went wrong!\n");
        printf("Error: %s\n",hipGetErrorString(error));

        clock_gettime(CLOCK_MONOTONIC, &finish);

        time_difference(&start,&finish,&difference);
        printf("Program run time: %lf\n", difference /1000000000.0 );

        exit(1);
    }
}

int main(){
    clock_gettime(CLOCK_MONOTONIC,&start);
    hipError_t error;
    
    
	char alphabetArrayHost[26] = {'a','b','c','d','e','f','g','h','i','j','k','l','m','n','o','p','q','r','s','t','u','v','w','x','y','z'};
	char numberArrayHost[10] = {'0','1','2','3','4','5','6','7','8','9'};

	char encryptedPassword[11] = "tpsmgi6291";

	char *decryptedPassword;
	decryptedPassword = (char *)malloc(sizeof(char) * 26);

	char * alphabetDataDevice;
	error = hipMalloc( (void**) &alphabetDataDevice, sizeof(char) * 26); 
    CheckForError(error);
    
	error = hipMemcpy(alphabetDataDevice, alphabetArrayHost, sizeof(char) * 26, hipMemcpyHostToDevice);
    CheckForError(error);

    char * numberDataDevice;
	hipMalloc( (void**) &numberDataDevice, sizeof(char) * 10); 
    CheckForError(error);

	hipMemcpy(numberDataDevice, numberArrayHost, sizeof(char) * 10, hipMemcpyHostToDevice);
    CheckForError(error);
	
    char *passwordToDecryptDevice;
	hipMalloc( (void**) &passwordToDecryptDevice, sizeof(char) * 26);
    CheckForError(error);

	hipMemcpy(passwordToDecryptDevice, encryptedPassword, sizeof(char) * 26, hipMemcpyHostToDevice);
    CheckForError(error);

    dim3 numberOfBlocksToUse = dim3(26,26,1); 
    dim3 threadsPerBlocks = dim3(10,10,1); 

    CrackPassword<<<numberOfBlocksToUse, threadsPerBlocks >>>( alphabetDataDevice, numberDataDevice, passwordToDecryptDevice );
	hipDeviceSynchronize();  
    
	hipMemcpy(decryptedPassword, passwordToDecryptDevice, sizeof(char) * 26, hipMemcpyDeviceToHost);
    CheckForError(error);

	printf("\nUser Given Encrypted Password: %s,\tDecrypted Password: %s\n\n", encryptedPassword, decryptedPassword);

    // free the pointer memory
	free(decryptedPassword);
	error = hipFree(alphabetDataDevice);
    CheckForError(error);

	error = hipFree(numberDataDevice);
    CheckForError(error);

	error = hipFree(passwordToDecryptDevice);
    CheckForError(error);

    clock_gettime(CLOCK_MONOTONIC, &finish);

    time_difference(&start,&finish,&difference);
    printf("Program run time: %lf\n\n", difference /1000000000.0 );

	return 0;
}

